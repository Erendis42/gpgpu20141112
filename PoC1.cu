
#include <hip/hip_runtime.h>
#include <stdio.h>

#define GRID 128 
#define BLOCK 32

//#define GRID 8
//#define BLOCK 2

__global__ void DumpMem(int* m)
{
	int i = blockIdx.x*gridDim.x+threadIdx.x;
	printf("%x ", m[i]);
}

__global__ void WriteToMem(int* m)
{
	int i = blockIdx.x*gridDim.x+threadIdx.x;
	m[i] = 0xC0FFEE43;
}

int main(int argc, char* argv[])
{
	hipSetDevice(0);

	int* dm;
	int* dn;

	int memSize = sizeof(int) * GRID * GRID * BLOCK * BLOCK;

	dim3 grid(GRID,GRID);
	dim3 block(BLOCK,BLOCK);

	hipMalloc(&dm, memSize);

	WriteToMem<<<grid,block>>>(dm);

	hipDeviceSynchronize();

	hipFree(dm);

	hipMalloc(&dn, memSize);

	DumpMem<<<grid,block>>>(dn);

	hipDeviceSynchronize();

	hipFree(dn);

	return 0;
}
